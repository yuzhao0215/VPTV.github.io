#include "hip/hip_runtime.h"

#ifndef CORRESPOND_CU_
#define CORRESPOND_CU_

#include "correspond.hpp"
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/copy.h>
#include <thrust/fill.h>
#include <thrust/sequence.h>
#include <thrust/functional.h>
#include <thrust/transform.h>

namespace pt {

using namespace std;

class Part {
public:
	float X[2];
	int id;
};

class DistanceCalc : public thrust::binary_function<Part&, Part&, float>
{
public:
	__host__ __device__
		float operator()(Part& A, Part& B) { 
			float dx = A.X[0]-B.X[0];
			float dy = A.X[1]-B.X[1];
			return sqrtf(dx*dx + dy*dy); 
	}
};


class EpipolarLineCalc : public thrust::binary_function<Part&, float**, float3>
{
public:
//__host__ __device__
	float3 operator()(Part& A, float** F) { 
		float x = A.X[0];
		float y = A.X[1];
		float z = 1.0;
		float3 line;
		line.x = F[0][0] * x + F[1][0] * y + F[2][0] * z;
		line.y = F[0][1] * x + F[1][1] * y + F[2][1] * z;
		line.z = F[0][2] * x + F[1][2] * y + F[2][2] * z;
		float factor = line.x * line.x + line.y * line.y;
		factor = factor ? 1./sqrt(factor) : 1.;
		line.x *= factor;
		line.y *= factor;
		line.z *= factor;
		return line;
	}
};

class EpipolarResidualCalc : public thrust::binary_function<Part&, float3&, float>
{
public:
	__host__ __device__
		float operator()(Part& A, float3& line) { 
			return fabs( A.X[0] * line.x + A.X[1] * line.y + line.z );
	}
};

void calcDistancesHOST(thrust::host_vector<Part>& A, thrust::host_vector<Part>& B, thrust::host_vector<float>& distances);
void calcDistancesDEVICE(thrust::device_vector<Part>& A, thrust::device_vector<Part>& B, thrust::device_vector<float>& distances);


} /* NAMESPACE_PT */

#endif /* CORRESPOND_CU_ */
